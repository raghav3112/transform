#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {
  
template <typename Dtype>
__global__ void log_ratio(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = log(max(in[index], Dtype(kLOG_THRESHOLD)));
  }
}

template <typename Dtype>
Dtype SoftmaxEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  // The forward pass computes the softmax prob values.
  softmax_bottom_vec_[0] = bottom[0];
  softmax_top_vec_[0] = &prob_;
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  
  const Dtype* prob_data = prob_.gpu_data();
  Dtype* log_data = log_vector_.mutable_gpu_data();
  int num = prob_.num();
  int count = prob_.count();
  
  log_ratio<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, prob_data, log_data);
  
  caffe_gpu_dot<Dtype>(count, log_data, prob_data, &loss_);
    
  return - min_val_ - loss_ * coeff_ / num;
//   return Forward_cpu(bottom, top);
}

template <typename Dtype>
void SoftmaxEntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  // Compute the diff
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* log_data = log_vector_.gpu_data();
  Dtype* ones_data = ones_.mutable_gpu_data();
  Dtype* e_persample_data = e_persample_.mutable_gpu_data();
  int num = prob_.num();
  int channels = prob_.channels();
  int count = prob_.count();
  
  caffe_gpu_mul<Dtype>(count, log_data, prob_data, bottom_diff);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels, channels, Dtype(1),
      bottom_diff, ones_data, Dtype(0), e_persample_data);
  caffe_gpu_mul<Dtype>(count, e_persample_data, prob_data, e_persample_data);
  caffe_gpu_axpy<Dtype>(count, -1, e_persample_data, bottom_diff);  
  caffe_gpu_scal(count, -coeff_ / Dtype(num), bottom_diff); 
}

INSTANTIATE_CLASS(SoftmaxEntropyLossLayer);


}  // namespace caffe

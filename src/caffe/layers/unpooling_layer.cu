#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolFixedForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, const int pad, Dtype* top_data,
    const int* mask, const Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride - pad;
    int wstart = pw * stride - pad;
    int hend = min(hstart + kernel_size, height);
    int wend = min(wstart + kernel_size, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    if (mask) {
      maxidx = mask[index];
    } else {
      maxidx = top_mask[index];
    }
    bottom_data += (n * channels + c) * height * width;
    maxval = bottom_data[maxidx];
    top_data[index] = maxval;
  }
}

template <typename Dtype>
__global__ void UnAvePoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, const int pad, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride - pad;
    int wstart = pw * stride - pad;
    int hend = min(hstart + kernel_size, height + pad);
    int wend = min(wstart + kernel_size, width + pad);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_data[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}


template <typename Dtype>
__global__ void UnMaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const Dtype* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_size, const int stride,
    const int pad, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad < kernel_size) ? 0 : (h + pad - kernel_size) / stride + 1;
    int phend = min((h + pad) / stride + 1, pooled_height);
    int pwstart =
        (w + pad < kernel_size) ? 0 : (w + pad - kernel_size) / stride + 1;
    int pwend = min((w + pad) / stride + 1, pooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    } else {
      top_mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (top_mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void UnAvePoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, const int pad,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_size) ? 0 : (h - kernel_size) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < kernel_size) ? 0 : (w - kernel_size) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride - pad;
        int wstart = pw * stride - pad;
        int hend = min(hstart + kernel_size, height + pad);
        int wend = min(wstart + kernel_size, width + pad);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
Dtype UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  int count = (*top)[0]->count();
  // we simply always assume that we have a mask 
  assert(top->size() > 1);
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    top_mask = bottom[1]->mutable_gpu_data();    
    // NOLINT_NEXT_LINE(whitespace/operators)
    UnMaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, top_mask, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_size_, stride_, pad_, top_data);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    UnAvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
        pad_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    LOG(FATAL) << "Not implemented.";
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
  return Dtype(0.);
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const int count = (*bottom)[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    assert(bottom->size() > 1);
    top_mask = (*bottom)[1]->gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolFixedForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_size_, stride_, pad_, bottom_diff, mask, top_mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    UnAvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
        pad_, bottom_diff);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:    
    LOG(FATAL) << "Not implemented.";
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_CLASS(UnPoolingLayer);


}  // namespace caffe

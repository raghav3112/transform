#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {
  
template <typename Dtype>
__global__ void log_ratio(const int n, const Dtype* in1, const Dtype* in2, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = log(max(in1[index], Dtype(FLT_MIN)))-log(max(in2[index], Dtype(FLT_MIN)));
  }
}

template <typename Dtype>
Dtype SoftmaxKLLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  // The forward pass computes the softmax prob values.
  softmax_bottom_vec_[0] = bottom[0];
  softmax_top_vec_[0] = &prob1_;
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  softmax_bottom_vec_[0] = bottom[1];
  softmax_top_vec_[0] = &prob2_;
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  
  const Dtype* prob1_data = prob1_.gpu_data();
  const Dtype* prob2_data = prob2_.gpu_data();
  Dtype* log_data = log_vector_.mutable_gpu_data();
  int num = prob1_.num();
  int count = prob1_.count();
  
  log_ratio<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, prob1_data, prob2_data, log_data);
  
  caffe_gpu_dot<Dtype>(count, log_data, prob1_data, &loss_);
  
  loss_ *= coeff_;
  
  return loss_ / num;
}

template <typename Dtype>
void SoftmaxKLLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  // Compute the diff
  Dtype* bottom_diff1 = (*bottom)[0]->mutable_gpu_diff();
  Dtype* bottom_diff2 = (*bottom)[1]->mutable_gpu_diff();
  const Dtype* prob1_data = prob1_.gpu_data();
  const Dtype* prob2_data = prob2_.gpu_data();
  const Dtype* log_data = log_vector_.gpu_data();
  int num = prob1_.num();
  int count = prob1_.count();  
  
  caffe_gpu_copy<Dtype>(count, prob2_data, bottom_diff2);
  caffe_gpu_axpy<Dtype>(count, Dtype(-1), prob1_data, bottom_diff2);  
  caffe_gpu_scal(count, coeff_ / num, bottom_diff2);
  
  caffe_gpu_mul<Dtype>(count, log_data, prob1_data, bottom_diff1);
  caffe_gpu_axpy<Dtype>(count, -loss_, prob1_data, bottom_diff1);  
  caffe_gpu_scal(count, coeff_ / num, bottom_diff1);
}

INSTANTIATE_CLASS(SoftmaxKLLossLayer);


}  // namespace caffe

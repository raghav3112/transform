#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {
  
template <typename Dtype>
__global__ void log_ratio(const int n, const Dtype* in1, const Dtype* in2, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = log(max(in1[index], Dtype(kLOG_THRESHOLD))) - log(max(in2[index], Dtype(kLOG_THRESHOLD)));
  }
}

template <typename Dtype>
Dtype SoftmaxKLLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  // The forward pass computes the softmax prob values.
  softmax_bottom_vec_[0] = bottom[0];
  softmax_top_vec_[0] = &prob1_;
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  softmax_bottom_vec_[0] = bottom[1];
  softmax_top_vec_[0] = &prob2_;
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  
  const Dtype* prob1_data = prob1_.gpu_data();
  const Dtype* prob2_data = prob2_.gpu_data();
  Dtype* log_data = log_vector_.mutable_gpu_data();
  int num = prob1_.num();
  int count = prob1_.count();
  
  log_ratio<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, prob1_data, prob2_data, log_data);
  
  caffe_gpu_dot<Dtype>(count, log_data, prob1_data, &loss_);
    
  return loss_ * coeff_ / num;
//   return Forward_cpu(bottom, top);
}

template <typename Dtype>
void SoftmaxKLLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  // Compute the diff
  Dtype* bottom_diff1 = (*bottom)[0]->mutable_gpu_diff();
  Dtype* bottom_diff2 = (*bottom)[1]->mutable_gpu_diff();
  const Dtype* prob1_data = prob1_.gpu_data();
  const Dtype* prob2_data = prob2_.gpu_data();
  const Dtype* log_data = log_vector_.gpu_data();
  Dtype* ones_data = ones_.mutable_gpu_data();
  Dtype* kl_persample_data = kl_persample_.mutable_gpu_data();
  int num = prob1_.num();
  int channels = prob1_.channels();
  int count = prob1_.count();
  
  caffe_gpu_mul<Dtype>(count, log_data, prob1_data, bottom_diff1);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels, channels, Dtype(1),
      bottom_diff1, ones_data, Dtype(0), kl_persample_data);
  caffe_gpu_mul<Dtype>(count, kl_persample_data, prob1_data, kl_persample_data);
  caffe_gpu_axpy<Dtype>(count, -1, kl_persample_data, bottom_diff1);  
  caffe_gpu_scal(count, coeff_ / Dtype(num), bottom_diff1); 
  
  caffe_gpu_copy<Dtype>(count, prob2_data, bottom_diff2);
  caffe_gpu_axpy<Dtype>(count, Dtype(-1), prob1_data, bottom_diff2);  
  caffe_gpu_scal(count, coeff_ / Dtype(num), bottom_diff2);
  
//   Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(SoftmaxKLLossLayer);


}  // namespace caffe

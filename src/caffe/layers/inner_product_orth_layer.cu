#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <hipblas.h>

#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
  
template <typename Dtype>
__global__ void make_nonneg(const int n, Dtype* in) {
  CUDA_KERNEL_LOOP(index, n) {
    in[index] = max(0., in[index]);      
  }
}

template <typename Dtype>
Dtype InnerProductOrthLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* weight = this->blobs_[0]->mutable_gpu_data();
  
  // Orthogonalizing
  if (Caffe::phase() == Caffe::TRAIN && orth_step_ > 0) {
    if (!(iter_ % orth_step_) && (orth_before_iter_ == 0 || iter_ < orth_before_iter_)) {
//       LOG(INFO) << "Orthogonalizing, iter=" << iter_;
      switch (orth_method_) {
        case OrthParameter_OrthMethod_ESMAEILI:
        {  
//           LOG(INFO) << "ESMAEILI";
          Dtype* gram = this->gram_.mutable_gpu_data();
          Dtype* kk = this->kk_.mutable_gpu_data();
          Dtype* ak = this->ak_.mutable_gpu_data();
          const Dtype* id = this->id_.gpu_data();
          Dtype error;

          caffe_gpu_scal(N_*K_, Dtype(1) / col_norm_, weight);
          for (int ni=0; ni<max_num_iter_; ni++) {
            caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, N_, N_, K_, (Dtype)1.,
                weight, weight, (Dtype)0., gram);
            caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, N_, N_, N_, esmaeili_coeff_,
                gram, gram, (Dtype)0., kk);
            caffe_gpu_axpy<Dtype>(N_*N_, -(1. + esmaeili_coeff_), gram, kk);
            caffe_gpu_axpy<Dtype>(N_*N_, Dtype(2), id, kk);
            caffe_gpu_axpy<Dtype>(N_*N_, Dtype(-1), id, gram);
            error = caffe_gpu_norm2(N_*N_, gram);
            //LOG(INFO) << "Iter " << ni+1 << "  ||Gram - id||=" << error;
            if (error < min_error_ + eps_)
              ni = max_num_iter_;
            else {
              LOG(INFO) << "Iter " << iter_ << "." << ni <<"  ||Gram - id||=" << error;
              caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, N_, K_, N_, (Dtype)1.,
                kk, weight, (Dtype)0., ak);
              caffe_gpu_copy(N_*K_, ak, weight);
            }
          }
          caffe_gpu_scal(N_*K_, col_norm_, weight);
          break;
        }
        case OrthParameter_OrthMethod_NORM_L2:
        {
          normalize_weights(min_norm_, max_norm_, target_norm_);
          break;
        }
        case OrthParameter_OrthMethod_NORM_L1:
        {
          normalize_weights_l1(min_norm_, max_norm_, target_norm_);
          break;
        }
        case OrthParameter_OrthMethod_NORM_L1_NONNEG:
        {
//           LOG(INFO) << "Make nonneg";
          make_nonneg<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
              this->blobs_[0]->count(), weight);
//           LOG(INFO) << "Normalize";
          normalize_weights_l1(min_norm_, max_norm_, target_norm_);
          break;
        }
        case OrthParameter_OrthMethod_NONE:
//           LOG(INFO) << "NONE";
          break;
        default:
          LOG(FATAL) << "Unknown orthogonalization method";
          break;  
      }
    }
    iter_++;
  }
  
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
      bottom_data, weight, (Dtype)0., top_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
        reinterpret_cast<const Dtype*>(bias_multiplier_->gpu_data()),
        this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }  
  
  return Dtype(0);
}

template <typename Dtype>
void InnerProductOrthLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  // Gradient with respect to weight
  caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
      top_diff, bottom_data, (Dtype)0., this->blobs_[0]->mutable_gpu_diff());
  if (bias_term_) {
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        reinterpret_cast<const Dtype*>(bias_multiplier_->gpu_data()),
        (Dtype)0., this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down) {
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff, this->blobs_[0]->gpu_data(), (Dtype)0.,
        (*bottom)[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_CLASS(InnerProductOrthLayer);

}  // namespace caffe

#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {
  
template <typename Dtype>
__global__ void neg_log(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = -log(max(in[index], Dtype(kLOG_THRESHOLD)));
  }
}

template <typename Dtype>
Dtype SoftmaxMultilabelLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  // The forward pass computes the softmax prob values.
  softmax_bottom_vec_[0] = bottom[0];
  softmax_layer_->Forward(softmax_bottom_vec_, &softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  Dtype* log_prob_data = log_prob_.mutable_gpu_data();
  Dtype* log_label_data = log_label_.mutable_gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  int num = prob_.num();
  int count = prob_.count();
  Dtype loss;
//   Dtype loss1;
  
  neg_log<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, prob_data, log_prob_data);
  neg_log<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, label, log_label_data);
  
  caffe_gpu_dot<Dtype>(count, label, log_prob_data, &loss);
//   caffe_gpu_dot<Dtype>(count, label, log_label_data, &loss1);     // comment out to make gradient tests work
//   loss -= loss1;                                                  // comment out to make gradient tests work   
  return loss / num;
}

template <typename Dtype>
void SoftmaxMultilabelLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  // Compute the diff
  Dtype* bottom_diff_net = (*bottom)[0]->mutable_gpu_diff();
  Dtype* bottom_diff_lbl = (*bottom)[1]->mutable_gpu_diff();
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* log_prob_data = log_prob_.gpu_data();
  int num = prob_.num();
  int count = prob_.count(); 
  
  CUDA_CHECK(hipMemcpy(bottom_diff_net, prob_data, sizeof(Dtype) * count, hipMemcpyDeviceToDevice));
//   memcpy(bottom_diff_net, prob_data, sizeof(Dtype) * count);
  const Dtype* label = (*bottom)[1]->gpu_data();  
  caffe_gpu_axpy<Dtype>(count, Dtype(-1), label, bottom_diff_net);
  // Scale down gradient
  caffe_gpu_scal(count, Dtype(1) / num, bottom_diff_net);
  
  CUDA_CHECK(hipMemcpy(bottom_diff_lbl, log_prob_data, sizeof(Dtype) * count, hipMemcpyDeviceToDevice));
//   memcpy(bottom_diff_lbl, log_prob_data, sizeof(Dtype) * count);
  caffe_gpu_scal(count, Dtype(1) / num, bottom_diff_lbl);
}

INSTANTIATE_CLASS(SoftmaxMultilabelLossLayer);


}  // namespace caffe

#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype InvertGradientLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  (*top)[0]->ShareData(*bottom[0]);
  return Dtype(0.);
}

template <typename Dtype>
void InvertGradientLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int count = top[0]->count();
  caffe_gpu_copy(count, top_diff, bottom_diff);
  caffe_gpu_scal(count, -coeff_, bottom_diff);
  
  iter_++;
  coeff_ = initial_coeff_ + (final_coeff_ - initial_coeff_) * (Dtype(2) / (Dtype(1) + exp(-gamma_ * iter_)) - Dtype(1));
}

INSTANTIATE_CLASS(InvertGradientLayer);

}  // namespace caffe
